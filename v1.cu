#include "hip/hip_runtime.h"
#include <math.h>
#include <stdio.h>
#include <stdlib.h>

#include "utils.h"
#include "v1.h"

/*
  TODO: Might be a good idea to implement a kernel for matrix padding
  instead of performing in on host
*/

// Define the kernel to calculate a moment per thread
__global__ void calc_moment(int *pad_in_matrix, int *pad_out_matrix, int size) {
  int i = blockIdx.x * blockDim.x + threadIdx.x;
  int j = blockIdx.y * blockDim.y + threadIdx.y;

  // Computation must not be performed on the border elements
  if (i < size - 1 && j < size - 1) {
    // calcualte moment and update out matrix
    pad_out_matrix[(i + 1) * size + (j + 1)] =
        calculate_moment(pad_in_matrix, i + 1, j + 1);
  }
}

void ising_model_v1(int *in_matrix, int *out_matrix, int model_size,
                          int num_iterations) {
  // Add appropriate padding to matrix to avoid checks on boundries
  int *pad_in_matrix = pad_matrix(in_matrix, model_size);
  int *out_matrix = (int *)calloc(model_size * model_size, sizeof(int));

  // Allocate memory for device copies
  int matrix_bytes = model_size * model_size * sizeof(int);
  int pad_matrix_bytes = (model_size + 1) * (model_size + 1) * sizeof(int);

  int *pad_in_matrix_d;
  int *out_matrix_d;

  hipMalloc((void **)&pad_in_matrix_d, pad_matrix_bytes);
  hipMalloc((void **)&out_matrix_d, matrix_bytes);

  // Copy data to device
  hipMemcpy(pad_in_matrix_d, pad_in_matrix, pad_matrix_bytes, hipMemcpyHostToDevice);
  hipMemcpy(out_matrix_d, out_matrix, matrix_bytes, hipMemcpyHostToDevice);

  // Calculate grid dimensions
  int BLOCK_SIZE = 32;  // So a block contains 1024 threads
  dim3 block_dim(BLOCK_SIZE, BLOCK_SIZE, 1);

  int GRID_SIZE = ceil(model_size / BLOCK_SIZE);
  dim3 grid_dim(GRID_SIZE, GRID_SIZE, 1);

  // THIS WILL BE KIND OF TRICKY (MAYBE theres no need for swap)
  // Matrix pad may be needed before launcing the kernel
  // for number of iterations,
  // Check weather in or out matrix contains the result and swap if needed

  // Copy data back from the device
  hipMemcpy(out_matrix, out_matrix_d, matrix_bytes, hipMemcpyDeviceToHost);

  // Device cleanup
  hipFree(pad_in_matrix_d);
  hipFree(out_matrix_d);
}
