#include "hip/hip_runtime.h"
#include <stdio.h>
#include <stdlib.h>

#include "utils.h"
#include "v2.h"

__device__ int calculate_moment_v2(int *matrix, int size, int i, int j) {
  int sign = matrix[(i - 1) * size + j] + matrix[(i + 1) * size + j] +
             matrix[i * size + j] + matrix[i * size + (j - 1)] +
             matrix[i * size + (j + 1)];

  return sign > 0 ? 1 : -1;
}
// Guess that's not the optimal implementation
// Might use v1 instead
__global__ void add_halo_v2(int *matrix, int size, int tile_width,
                            int *pad_matrix) {
  int row_start = blockIdx.y * tile_width;
  int row_end = row_start + tile_width;
  int col_start = blockIdx.x * tile_width;
  int col_end = col_start + tile_width;

  for (int i = row_start; i < row_end; i++) {
    for (int j = col_start; j < col_end; j++) {
      if (i < size && j < size) {
        // Copy elements from matrix to padded matrix
        pad_matrix[(i + 1) * (size + 2) + j + 1] = matrix[i * size + j];

        if (j == 0) {
          // Top Padding
          pad_matrix[i + 1] = matrix[(size - 1) * size + i];
          // Right padding
          pad_matrix[(i + 1) * (size + 2) + (i + 1)] = matrix[i * size];
          // Bottom padding
          pad_matrix[(size + 1) * (size + 2) + (i + 1)] = matrix[i];
          // Left padding
          pad[(i + 1) * (size + 2)] = matrix[i * size + (size - 1)];
        }
      }
    }
  }
}

__global__ void update_model_v2(int *pad_in_matrix, int *out_matrix, int size,
                                int tile_width) {
  int row_start = (blockIdx.y * blockDim.y + threadIdx.y) * tile_width;
  int row_end = row_start + tile_width;
  int col_start = (blockIdx.x * blockDim.x + threadIdx.x) * tile_width;
  int col_end = col_start + tile_width;

  for (int i = row_start; i < row_end; i++)
    for (int j = col_start; j < col_end; j++)
      if (i < size && j < size)
        out_matrix[i * size + j] =
            calculate_moment_v2(pad_in_matrix, size + 2, i + 1, j + 1);
}

// A thread calculates a tile of moments
int *ising_model_v2(int *in_matrix, int size, int tile_width,
                    int num_iterations) {
  int *out_matrix = (int *)malloc(size * size, sizeof(int));

  // Allocate memory for device copies
  int matrix_bytes = size * model_size * sizeof(int);
  int pad_matrix_bytes = (size + 2) * (size + 2) * sizeof(int);

  int *in_matrix_d;
  int *pad_in_matrix_d;
  int *out_matrix_d;

  hipMalloc((void **)&in_matrix_d, matrix_bytes);
  hipMalloc((void **)&pad_in_matrix_d, pad_matrix_bytes);
  hipMalloc((void **)&out_matrix_d, matrix_bytes);

  // Copy data to device
  hipMemcpy(in_matrix_d, in_matrix, matrix_bytes, hipMemcpyHostToDevice);

  // Calculate block and grid dimensions
  dim3 block_dim(1, 1);

  int GRID_SIZE = (size + tile_width - 1) / tile_width;
  dim3 grid_dim(GRID_SIZE, GRID_SIZE);

  int k = 0;
  while (k < num_iterations) {
    add_halo_v2<<<grid_dim, block_dim>>>(in_matrix_d, size, tile_width,
                                         pad_in_matrix_d);

    update_model_v2<<<grid_dim, block_dim>>>(pad_in_matrix_d, out_matrix_d,
                                             size, tile_width);

    swap_matrices(&in_matrix_d, &out_matrix_d);
    k++;
  }

  hipMemcpy(out_matrix, in_matrix_d, matrix_bytes, hipMemcpyDeviceToHost);

  hipFree(in_matrix_d);
  hipFree(pad_in_matrix_d);
  hipFree(out_matrix_d);

  return out_matrix;
}
