#include "hip/hip_runtime.h"
#include <stdio.h>
#include <stdlib.h>

#include "v2.h"
#include "utils.h"

__global__ void update_model_v2(int *pad_in_matrix, int *out_matrix, int size, int tile_width) {
  int row_start = (blockIdx.y * blockDim.y + threadIdx.y) * tile_width;
  int row_end = row_start + tile_width;
  int col_start = (blockIdx.x * blockDim.x + threadIdx.x) * tile_width;
  int col_end = col_start + tile_width;

    for (int i = row_start; i < row_end; i++)
      for (int j = col_start; j < col_end; j++)
        if (i < size && j < size)
          out_matrix[i * size + j] = calculate_moment(pad_in_matrix, size + 2, i + 1, j + 1);
}

// A thread calculates a tile of moments
int *ising_model_v2(int *in_matrix, int size, int tile_width,
                    int num_iterations) {
  int *out_matrix = (int *)malloc(size * size, sizeof(int));

  // Allocate memory for device copies
  int matrix_bytes = size * model_size * sizeof(int);
  int pad_matrix_bytes = (size + 2) * (size + 2) * sizeof(int);

  int *in_matrix_d;
  int *pad_in_matrix_d;
  int *out_matrix_d;

  hipMalloc((void **)&in_matrix_d, matrix_bytes);
  hipMalloc((void **)&pad_in_matrix_d, pad_matrix_bytes);
  hipMalloc((void **)&out_matrix_d, matrix_bytes);

  // Copy data to device
  hipMemcpy(in_matrix_d, in_matrix, matrix_bytes, hipMemcpyHostToDevice);

  // Calculate block and grid dimensions
  dim3 block_dim(1, 1);

  int GRID_SIZE = (size + tile_width - 1) / tile_width;
  dim3 grid_dim(GRID_SIZE, GRID_SIZE);

  int k = 0;
  while (k < num_iterations) {
    // Add halo to matrix
      // (Dont know how to launch the previous kernel now)
    update_model_v2<<<grid_dim, block_dim>>>(pad_in_matrix_d, out_matrix_d, size);

    swap_matrices(&in_matrix_d, &out_matrix_d);
    k++;
  }

  hipMemcpy(out_matrix, in_matrix_d, matrix_bytes, hipMemcpyDeviceToHost);

  hipFree(in_matrix_d);
  hipFree(pad_in_matrix_d);
  hipFree(out_matrix_d);

  return out_matrix;
}
