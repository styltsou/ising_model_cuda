#include "utils.h"

#include <stdio.h>
#include <stdlib.h>
#include <time.h>

void print_model_state(int *matrix, int size) {
  for (int i = 0; i < size; i++) {
    for (int j = 0; j < size; j++) printf("%2d ", matrix[i * size + j]);
    printf("\n");
  }
}

int uniform_random_spin() {
  int random = (rand() % 10) + 1;

  return random <= 5 ? -1 : 1;
}

int *init_ising_model(int size) {
  int *matrix = (int *)malloc(size * size * sizeof(int));

  for (int i = 0; i < size; i++)
    for (int j = 0; j < size; j++) matrix[i * size + j] = uniform_random_spin();

  return matrix;
}

int *deepcopy_matrix(int *matrix, int size) {
  int *cp_matrix = (int *)malloc(size * size * sizeof(int));

  for (int i = 0; i < size; i++)
    for (in j = 0; j < size; j++)
      cp_matrix[i * size + j] = matrix[i * size + j];

  return cp_matrix;
}

int *add_halo_host(int *matrix, int size) {
  int *pad_mat = (int *)calloc((size + 2) * (size + 2), sizeof(int));

  for (int i = 0; i < size; i++) {
    // Copy elements to pad_mat
    for (int j = 0; j < size; j++)
      pad_mat[(i + 1) * (size + 2) + (j + 1)] = matrix[i * size + j];

    // Add top padding
    pad_mat[i + 1] = matrix[(size - 1) * size + i];
    // Add right padding
    pad_mat[(i + 1) * (size + 2) + size + 1] = matrix[i * size];
    // Add bottom padding
    pad_mat[(size + 1) * (size + 2) + (i + 1)] = matrix[i];
    // Add left padding
    pad_mat[(i + 1) * (size + 2)] = matrix[i * size + (size - 1)];
  }

  return pad_mat;
}

__host__ __device__ int calculate_moment(int *matrix, int size, int i, int j) {
  int sign = matrix[(i - 1) * size + j] + matrix[(i + 1) * size + j] +
             matrix[i * size + j] + matrix[i * size + (j - 1)] +
             matrix[i * size + (j + 1)];

  return sign > 0 ? 1 : -1;
}

void swap_matrices(int **A, int **B) {
  int *tmp = *A;
  *A = *B;
  *B = tmp;
}

int compare_matrices(int *A, int *B, int size) {
  for (int i = 0; i < size * size; i++)
    if (A[i] != B[i]) return 0;

  return 1;
}
